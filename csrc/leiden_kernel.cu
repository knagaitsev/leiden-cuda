#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add_kernel(float* a, float* b, float* c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] + b[idx];
}

extern "C" void launch_add_kernel(float* a, float* b, float* c, int N) {
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    add_kernel<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}
