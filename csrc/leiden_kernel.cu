#include <hip/hip_runtime.h>
#include <iostream>

typedef struct node_data {
    uint32_t community;
    uint32_t move_candidate;
    uint32_t agg_count;
} node_data_t;

typedef struct comm_data {
    uint32_t agg_count;
} comm_data_t;

__global__ void add_kernel(float* a, float* b, float* c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] + b[idx];
}

extern "C" void launch_add_kernel(float* a, float* b, float* c, int N) {
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    add_kernel<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

// parallelized at node level
// - we should also try parallelizing at edge level
__global__ void gather_move_candidates_kernel(uint32_t *offsets, uint32_t *indices, float *weights, node_data_t *node_data, comm_data_t *comm_data, int vertex_count, int edge_count, int comm_count, float gamma) {
    unsigned int node = threadIdx.x;
    
    // communities[threadIdx.x] = 1;
    uint32_t offset = offsets[node];
    uint32_t offset_next = offsets[node + 1];

    uint32_t curr_comm = node_data[node].community;

    uint32_t best_comm = curr_comm;
    float best_delta = 0.0f;

    uint32_t node_edge_count = offset_next - offset;

    // aggregate count of nodes in old community (including current node)
    int agg_count_old = comm_data[curr_comm].agg_count;

    // aggregate count of current node
    int node_agg_count = node_data[node].agg_count;

    // total edge weight of incoming edges from old community
    float k_vc_old = 0.0;
    for (uint32_t i = offset; i < offset_next; i++) {
        uint32_t neigh = indices[i];
        if (node_data[neigh].community == curr_comm) {
            k_vc_old += weights[i];
        }
    }

    for (uint32_t i = offset; i < offset_next; i++) {
        uint32_t neighbor = indices[i];
        float weight = weights[i];

        uint32_t neighbor_comm = node_data[neighbor].community;

        if (neighbor_comm == curr_comm || neighbor_comm == best_comm) {
            continue;
        }

        // aggregate count of nodes in new community (excluding current node)
        int agg_count_new = comm_data[neighbor_comm].agg_count;

        // total edge weight of incoming edges from new community
        float k_vc_new = 0.0;
        // TODO: need to try moving this elsewhere
        for (uint32_t j = offset; j < offset_next; j++) {
            uint32_t neigh = indices[j];
            if (node_data[neigh].community == neighbor_comm) {
                k_vc_new += weights[j];
            }
        }

        float delta = (k_vc_new - gamma * (float)(node_agg_count * agg_count_new)) - (k_vc_old - gamma * (float)(node_agg_count * (agg_count_old - node_agg_count)));

        if (delta > best_delta) {
            // printf("Node: %d, Delta: %f, best_comm: %d\n", node, delta, best_comm);
            best_delta = delta;
            best_comm = neighbor_comm;
        }
    }

    if (best_comm != curr_comm) {
        node_data[node].move_candidate = best_comm;
    }
}

// two approaches to doing move_nodes_fast: parallelizing at node level is below
// - another option is parallelizing at edge level, letting each thread consider an edge
__global__ void move_nodes_fast_kernel(uint32_t *offsets, uint32_t *indices, float *weights, node_data_t *node_data, comm_data_t *comm_data, int vertex_count, int edge_count, int comm_count, float gamma) {
    
}

template <typename T>
T* allocate_and_copy_to_device(T* data_host, int len) {
    T* data_device;

    int size = len * sizeof(T);
    hipMalloc((void**)&data_device, size);
    hipMemcpy(data_device, data_host, size, hipMemcpyHostToDevice);

    return data_device;
}

template <typename T>
T* copy_from_device(T* data_host, T* data_device, int len) {
    int size = len * sizeof(T);
    hipMemcpy(data_host, data_device, size, hipMemcpyDeviceToHost);
}

extern "C" void move_nodes_fast(uint32_t *offsets, uint32_t *indices, float *weights, node_data_t *node_data, comm_data_t *comm_data, int vertex_count, int edge_count, int comm_count, float gamma) {
    // each thread of the cuda kernel considers one node and attempts to greedily increase the CPM
    // by moving it to the best neighboring community

    // the threads should be considering nodes in a semi-random order though.
    // will it be better to give them an array of random indices to access,
    // - or should we reorder the data structure to ensure a warp coalesces global memory accesses?

    uint32_t *offsets_device = allocate_and_copy_to_device(offsets, vertex_count + 1);
    uint32_t *indices_device = allocate_and_copy_to_device(indices, edge_count);
    float *weights_device = allocate_and_copy_to_device(weights, edge_count);
    node_data_t *node_data_device = allocate_and_copy_to_device(node_data, vertex_count);
    comm_data_t *comm_data_device = allocate_and_copy_to_device(comm_data, comm_count);

    dim3 dim_grid(1);
 	dim3 dim_block(vertex_count);

    gather_move_candidates_kernel <<<dim_grid, dim_block>>> (offsets_device, indices_device, weights_device, node_data_device, comm_data_device, vertex_count, edge_count, comm_count, gamma);

	// move_nodes_fast_kernel <<<dim_grid, dim_block>>> (offsets_device, indices_device, weights_device, node_data_device, comm_data_device, vertex_count, edge_count, comm_count, gamma);

    hipDeviceSynchronize();

    copy_from_device(offsets, offsets_device, vertex_count + 1);
    copy_from_device(indices, indices_device, edge_count);
    copy_from_device(weights, weights_device, edge_count);
    copy_from_device(node_data, node_data_device, vertex_count);
    copy_from_device(comm_data, comm_data_device, comm_count);

    hipFree(offsets_device);
    hipFree(indices_device);
    hipFree(weights_device);
    hipFree(node_data_device);
    hipFree(comm_data_device);
}
